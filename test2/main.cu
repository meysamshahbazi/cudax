#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void hello_cuda()
{
    printf("Hello Cuda world \n");
}

__global__ void uniqe_idx_calc_threadIx( int * input)
{
    int tid = threadIdx.x;
    printf("threadIDX: %d, vlaue : %d \n", tid, input[tid]);
}


__global__ void uniqe_gid_calculation( int * input)
{
    int tid = threadIdx.x;
    int offset = blockIdx.x * blockDim.x;
    int gid = tid + offset;

    printf("blockIdx.x : %d, threadIdx.x : %d, gid : %d, value : %d \n",
		blockIdx.x, tid, gid, input[gid]);

}

__global__ void uniqe_gid_calculation_2d( int * data)
{
    int tid = threadIdx.x;
    int block_offset = blockDim.x* blockIdx.x;
    int row_offset = blockDim.x*gridDim.x*blockIdx.y;

    int gid = row_offset+block_offset+tid;

    printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, gid : %d - data : %d \n",
		blockIdx.x, blockIdx.y, tid, gid, data[gid]);
}

__global__ void uniqe_gid_calculation_2d_2d( int * data)
{
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    int num_thread_in_block = blockDim.x + blockDim.y;

    int block_offset =  blockIdx.x * num_thread_in_block;

    int num_thread_row = num_thread_in_block*gridDim.x;
    
    int row_offset = num_thread_row*blockIdx.y;

    int gid = row_offset+block_offset+tid;

    printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, gid : %d - data : %d \n",
		blockIdx.x, blockIdx.y, tid, gid, data[gid]);
}


int main()
{
    dim3 block(4);
    dim3 grid( 8);
    // hello_cuda<<<grid,block>>>();
    hipDeviceSynchronize();
    hipDeviceReset();

    int array_size = 16;
    int array_byte_size = sizeof(int) * array_size;
    int h_data[] = {23,9,4,53,65,12,1,33,87,45,23,12,342,56,44,99};

    for (int i = 0; i<array_size; i++ )
    {
        printf("%d ", h_data[i]);
    }
    printf("\n \n");

    int * d_data;

    hipMalloc( (void **) &d_data, array_byte_size );
    hipMemcpy( d_data,h_data ,array_byte_size , hipMemcpyHostToDevice);

    block = 16;
    grid = 1;

    
    uniqe_idx_calc_threadIx<<<grid,block >>>(d_data);
    
    hipDeviceSynchronize();
    printf("\n \n");
    block = 4;
    grid = dim3(2,2);
    uniqe_gid_calculation<<<grid,block >>>(d_data);


    hipDeviceSynchronize();
    block = 4;
	grid = dim3(2,2);

	uniqe_gid_calculation_2d << < grid, block >> > (d_data);
	hipDeviceSynchronize();

    block = dim3(2,2);
	grid = dim3(2,2);
    printf("---------------------------------------------------- \n"); 
	uniqe_gid_calculation_2d_2d << < grid, block >> > (d_data);
	hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}

