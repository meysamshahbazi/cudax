#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include <stdlib.h>
#include <cstring>
#include <time.h>

__global__ void hello_cuda()
{
    printf("Hello Cuda world \n");
}

__global__ void uniqe_idx_calc_threadIx( int * input)
{
    int tid = threadIdx.x;
    printf("threadIDX: %d, vlaue : %d \n", tid, input[tid]);
}


__global__ void uniqe_gid_calculation( int * input)
{
    int tid = threadIdx.x;
    int offset = blockIdx.x * blockDim.x;
    int gid = tid + offset;

    printf("blockIdx.x : %d, threadIdx.x : %d, gid : %d, value : %d \n",
		blockIdx.x, tid, gid, input[gid]);

}

__global__ void uniqe_gid_calculation_2d( int * data)
{
    int tid = threadIdx.x;
    int block_offset = blockDim.x* blockIdx.x;
    int row_offset = blockDim.x*gridDim.x*blockIdx.y;

    int gid = row_offset+block_offset+tid;

    printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, gid : %d - data : %d \n",
		blockIdx.x, blockIdx.y, tid, gid, data[gid]);
}

__global__ void uniqe_gid_calculation_2d_2d( int * data)
{
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    int num_thread_in_block = blockDim.x + blockDim.y;

    int block_offset =  blockIdx.x * num_thread_in_block;

    int num_thread_row = num_thread_in_block*gridDim.x;
    
    int row_offset = num_thread_row*blockIdx.y;

    int gid = row_offset+block_offset+tid;

    printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, gid : %d - data : %d \n",
		blockIdx.x, blockIdx.y, tid, gid, data[gid]);
}

__global__ void mem_trs_test(int *input)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
	printf("tid : %d , gid : %d, value : %d \n",threadIdx.x,gid,input[gid]);

}

__global__ void mem_trs_test2(int *input,int size )
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < size)
	    printf("tid : %d , gid : %d, value : %d \n",threadIdx.x,gid,input[gid]);

}

void query_device()
{
	int deviceCount = 0;   
	hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0)
	{
		printf("No CUDA support device found");
	}

	int devNo = 0;
	hipDeviceProp_t iProp; 
	hipGetDeviceProperties(&iProp, devNo);

	printf("Device %d: %s\n", devNo, iProp.name);
	printf("  Number of multiprocessors:                     %d\n",
		iProp.multiProcessorCount);
	printf("  clock rate :                     %d\n",
		iProp.clockRate);
	printf("  Compute capability       :                     %d.%d\n",
		iProp.major, iProp.minor);
	printf("  Total amount of global memory:                 %4.2f KB\n",
		iProp.totalGlobalMem / 1024.0);
	printf("  Total amount of constant memory:               %4.2f KB\n",
		iProp.totalConstMem / 1024.0);
	printf("  Total amount of shared memory per block:       %4.2f KB\n",
		iProp.sharedMemPerBlock / 1024.0);
	printf("  Total amount of shared memory per MP:          %4.2f KB\n",
		iProp.sharedMemPerMultiprocessor / 1024.0);
	printf("  Total number of registers available per block: %d\n",
		iProp.regsPerBlock);
	printf("  Warp size:                                     %d\n",
		iProp.warpSize);
	printf("  Maximum number of threads per block:           %d\n",
		iProp.maxThreadsPerBlock);
	printf("  Maximum number of threads per multiprocessor:  %d\n",
		iProp.maxThreadsPerMultiProcessor);
	printf("  Maximum number of warps per multiprocessor:    %d\n",
		iProp.maxThreadsPerMultiProcessor / 32);
	printf("  Maximum Grid size                         :    (%d,%d,%d)\n",
		iProp.maxGridSize[0], iProp.maxGridSize[1], iProp.maxGridSize[2]);
	printf("  Maximum block dimension                   :    (%d,%d,%d)\n",
		iProp.maxThreadsDim[0], iProp.maxThreadsDim[1], iProp.maxThreadsDim[2]);
}


int main()
{
    dim3 block(4);
    dim3 grid( 8);
    // hello_cuda<<<grid,block>>>();
    hipDeviceSynchronize();
    hipDeviceReset();

    int array_size = 16;
    int array_byte_size = sizeof(int) * array_size;
    int h_data[] = {23,9,4,53,65,12,1,33,87,45,23,12,342,56,44,99};

    for (int i = 0; i<array_size; i++ )
    {
        printf("%d ", h_data[i]);
    }
    printf("\n \n");

    int * d_data;

    hipMalloc( (void **) &d_data, array_byte_size );
    hipMemcpy( d_data,h_data ,array_byte_size , hipMemcpyHostToDevice);

    block = 16;
    grid = 1;

    
    uniqe_idx_calc_threadIx<<<grid,block >>>(d_data);
    
    hipDeviceSynchronize();
    printf("\n \n");
    block = 4;
    grid = dim3(2,2);
    uniqe_gid_calculation<<<grid,block >>>(d_data);


    hipDeviceSynchronize();
    block = 4;
	grid = dim3(2,2);

	uniqe_gid_calculation_2d << < grid, block >> > (d_data);
	hipDeviceSynchronize();

    block = dim3(2,2);
	grid = dim3(2,2);
    printf("----------------------------------------------------\n"); 
	uniqe_gid_calculation_2d_2d << < grid, block >> > (d_data);
	hipDeviceSynchronize();


    printf("----------------------------------------------------\n"); 
    int size = 150;
    int byte_size = size*sizeof(int);

    int * h_input; // host varible 
    h_input = (int *)malloc(byte_size);

    time_t t;
    srand( (unsigned int ) time(&t));
    for (int i = 0; i < size ; i++)
    {
        h_input[i] = (int) (rand() & 0xff);
        printf(" %d ,",h_input[i]);
    }


    printf("\n----------------------------------------------------\n"); 
    int *d_input;
    hipMalloc((void **) &d_input,byte_size );

    hipMemcpy(d_input,h_input,byte_size,hipMemcpyHostToDevice);

    block = 32;
    grid = 5;

    mem_trs_test<<<grid,block>>>(d_input);
    hipDeviceSynchronize();
    printf("\n----------------------------------------------------\n");
    block = 32;
    grid = 5;

    mem_trs_test2<<<grid,block>>>(d_input,size);
    hipDeviceSynchronize();


    hipFree(d_input);
    free(h_input);
    hipDeviceReset();
    query_device();
    return 0;
}

